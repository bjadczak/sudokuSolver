#include "../headers/solver.cuh"

__host__ int solveSudoku(int *start_board)
{
    hipError_t cudaStatus;
    int *sudokuBoard = 0;
    int tmpSudokuBoard[CELL_COUNT * NUM_OF_THREADS];
    possibleBoard *poss_d = 0, *poss_h = 0;

    poss_h = new possibleBoard[BOARD_SIZE * NUM_OF_THREADS];

    std::priority_queue<possibleBoard, std::vector<possibleBoard>, decltype(cmpQueue)> S(cmpQueue);

    try
    {

        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess)
        {
            std::ostringstream errMess;
            errMess << "hipSetDevice failed! Do you have a CUDA-capable GPU installed? Returned error code " << cudaStatus;
            throw sudokuSolverException(errMess.str());
        }

        cudaStatus = hipMalloc((void **)&sudokuBoard, NUM_OF_THREADS * CELL_COUNT * sizeof(int));
        if (cudaStatus != hipSuccess)
        {
            std::ostringstream errMess;
            errMess << "hipMalloc failed! Returned error code " << cudaStatus;
            throw sudokuSolverException(errMess.str());
        }

        cudaStatus = hipMalloc((void **)&poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard));
        if (cudaStatus != hipSuccess)
        {
            std::ostringstream errMess;
            errMess << "hipMalloc failed! Returned error code " << cudaStatus;
            throw sudokuSolverException(errMess.str());
        }

        cudaStatus = hipMemcpy(sudokuBoard, start_board, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            std::ostringstream errMess;
            errMess << "hipMemcpy failed! Returned error code " << cudaStatus;
            throw sudokuSolverException(errMess.str());
        }

        // We run 1 Kernel for our start board, this will give us at least 1 (if board is not finished on start)
        // up to 9 boards. For those we run the solver further
        printBoard((int *)start_board);
        runSolver<<<1, 1>>>(sudokuBoard, poss_d);

        fetchResoults(cudaStatus, poss_h, poss_d);

        // Store the possible boards in priority queue
        for (int i = 0; i < NUM_OF_THREADS; i++)
        {
            for (int j = 0; j < BOARD_SIZE; j++)
            {
                if (poss_h[i * BOARD_SIZE + j].status >= 1)
                {
#ifdef DEBUG_MODE
                    printf("Possible board [THREAD: %d][POSS: %d]:\n", i + 1, j + 1);
                    printBoard(poss_h[i * BOARD_SIZE + j].board);
#endif
                    possibleBoard tmp;
                    for (int k = 0; k < CELL_COUNT; k++)
                        tmp.board[k] = poss_h[i * BOARD_SIZE + j].board[k];
                    tmp.status = poss_h[i * BOARD_SIZE + j].status;
                    S.push(tmp);
                }
            }
        }

        // Until there are boards to be checked
        while (!S.empty())
        {
            // Input new boards
            int indx = 0;
#ifdef DEBUG_MODE
            printf("%ld\n", S.size());
#endif
            // Fetch as many boards as we have Threads avaliable
            for (; indx < NUM_OF_THREADS && !S.empty(); indx++)
            {
                possibleBoard tmp = S.top();
                S.pop();
#ifdef DEBUG_MODE
                printf("Running thread %02d with board:\n", indx + 1);
                printBoard(tmp.board);
#endif
                // Write down the boards to GPU memory
                for (int j = 0; j < CELL_COUNT; j++)
                {
                    tmpSudokuBoard[indx * CELL_COUNT + j] = tmp.board[j];
                }
            }

            // Copy memory and run kernel
            cudaStatus = hipMemcpy(sudokuBoard, tmpSudokuBoard, (NUM_OF_THREADS)*CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
            {
                std::ostringstream errMess;
                errMess << "hipMemcpy failed! Returned error code " << cudaStatus;
                throw sudokuSolverException(errMess.str());
            }
#ifdef DEBUG_MODE
            printf("Running %02d threads\n", indx);
#endif
            // Run Kernels
            runSolver<<<1, indx>>>(sudokuBoard, poss_d);

            // Fetch resoults
            fetchResoults(cudaStatus, poss_h, poss_d);

            // Add new boards to S
            int *tmpBoard;
            if ((tmpBoard = addNewBoardsToQueue(indx, poss_h, S)))
            {
                start_board = tmpBoard;
                break;
            }
        }

        if (isBoardValid(start_board))
        {
            printf("SOLVED!\n");
            printBoard(start_board);
        }
        else
        {
            printf("BOARD UNSOLVABLE!\n");
        }
    }
    catch (sudokuSolverException &e)
    {
        fprintf(stderr, "%s", e.what());
        hipFree(sudokuBoard);
        hipFree(poss_d);
        delete[] poss_h;

        return cudaStatus;
    }

    hipFree(sudokuBoard);
    hipFree(poss_d);
    delete[] poss_h;

    return cudaStatus;
}

__host__ int *addNewBoardsToQueue(int &indx, possibleBoard *poss_h, std::priority_queue<possibleBoard, std::vector<possibleBoard>, decltype(cmpQueue)> &Q)
{
    for (int i = 0; i < indx; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (poss_h[i * BOARD_SIZE + j].status >= 1)
            {
#ifdef DEBUG_MODE
                printf("Possible board [THREAD: %d][POSS: %d]:\n", i + 1, j + 1);
                printBoard(poss_h[i * BOARD_SIZE + j].board);
#endif
                if (isBoardValid(poss_h[i * BOARD_SIZE + j].board))
                {
                    return poss_h[i * BOARD_SIZE + j].board;
                }

                possibleBoard tmp;
                for (int k = 0; k < CELL_COUNT; k++)
                    tmp.board[k] = poss_h[i * BOARD_SIZE + j].board[k];
                tmp.status = poss_h[i * BOARD_SIZE + j].status;
                Q.push(tmp);
            }
        }
    }

    return nullptr;
}
__host__ void fetchResoults(hipError_t &cudaStatus, possibleBoard *poss_h, possibleBoard *poss_d) throw()
{
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        std::ostringstream errMess;
        errMess << "Kernel launch failed: " << hipGetErrorString(cudaStatus);
        throw sudokuSolverException(errMess.str());
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        std::ostringstream errMess;
        errMess << "hipDeviceSynchronize returned error code " << cudaStatus;
        throw sudokuSolverException(errMess.str());
    }

    cudaStatus = hipMemcpy(poss_h, poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        std::ostringstream errMess;
        errMess << "hipMemcpy failed! Returned error code " << cudaStatus;
        throw sudokuSolverException(errMess.str());
    }
}