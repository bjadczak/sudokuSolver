#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define N 3
#define BOARD_SIZE N*N
#define CELL_COUNT BOARD_SIZE*BOARD_SIZE

#define DEBUG_MODE

__device__ void printBoard(int* sudokuBoard, int* targetCell, int id)
{
    for(int i = 0; i < BOARD_SIZE; i++)
    {
        for(int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d ", sudokuBoard[i*BOARD_SIZE + j]);
        }
        printf("\n")
    }
    printf("TARGET CELL - %d\n", *targetCell);
}

__global__ void fillEmpty(int* sudokuBoard, int* targetCell)
{
    #ifdef DEBUG_MODE
        printBoard(sudokuBoard, targetCell, threadIdx.x);
    #endif

    // Tables we use to count appearence
    int appeardInRow[BOARD_SIZE];
    int appeardInColumn[BOARD_SIZE];
    int appeardInBlock[BOARD_SIZE];

}

int main()
{
   hipError_t cudaStatus;

   const int start_board[CELL_COUNT] =
        {
            3, 0, 0, 8, 0, 1, 0, 0, 2,
            2, 0, 1, 0, 3, 0, 6, 0, 4,
            0, 0, 0, 0, 1, 0, 0, 0, 0,
            8, 0, 9, 0, 0, 0, 1, 0, 6,
            0, 6, 0, 0, 0, 0, 0, 5, 0,
            7, 0, 2, 0, 0, 0, 4, 0, 9,
            0, 0, 0, 5, 0, 9, 0, 0, 0,
            9, 0, 4, 0, 8, 0, 7, 0, 5,
            6, 0, 0, 0, 0, 7, 0, 0, 3,
        };

    int empty_cells[CELL_COUNT] = { -1 };

    int indx = 0;
    for(int i = 0; i < CELL_COUNT; i++)
        if(start_board[i] == 0)
        {
            empty_cells[indx] = i;
            indx++;
        }

   int* sudokuBoard = 0;
   int* targetCell = 0;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&sudokuBoard, CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&targetCell, CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(sudokuBoard, start_board, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(targetCell, empty_cells, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    fillEmpty<<<1, 1>>>(sudokuBoard, targetCell);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    hipFree(sudokuBoard);
    
    return cudaStatus;
}