#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

#include <stack>
#include <queue>
#include <chrono>

#define N 3
#define BOARD_SIZE (N * N)
#define CELL_COUNT (BOARD_SIZE * BOARD_SIZE)
#define NUM_OF_THREADS 1024
#define NUM_OF_BLOCKS 10
#define NUM_OF_KERNELS (NUM_OF_THREADS * NUM_OF_BLOCKS)

//#define DEBUG_MODE

struct appeared
{
    int cell = -1;

    int appeardInRow[BOARD_SIZE] = {0};
    int appeardInColumn[BOARD_SIZE] = {0};
    int appeardInBlock[BOARD_SIZE] = {0};

} typedef appeared;

struct possibilitie
{
    int cell = -1;
    int poss[BOARD_SIZE] = {0};

} typedef possibilitie;
struct possibleBoard
{
    int board[CELL_COUNT] = {0};
    int status = 0;

} typedef possibleBoard;
struct board
{
    int board[CELL_COUNT] = {0};

} typedef board;

struct move
{
    int board[CELL_COUNT] = {0};
    int cell = -1;
    int possibilites[BOARD_SIZE] = {0};

    move(int *currentBoard, int cell, int *possibilites)
    {
        for (int i = 0; i < CELL_COUNT; i++)
            this->board[i] = currentBoard[i];

        for (int i = 0; i < BOARD_SIZE; i++)
            this->possibilites[i] = possibilites[i];

        this->cell = cell;
    }
} typedef move;

__global__ void
fillEmpty(const int *sudokuBoard, const int *targetCell, appeared *app);

__host__ void printBoardInfo(appeared *app)
{
    for (int i = 0; i < CELL_COUNT; i++)
    {

        printf("[%d]TARGET CELL - %d\n", i, app[i].cell);

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInRow[j]);
        }
        printf(" - Appeared in row\n");

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInColumn[j]);
        }
        printf(" - Appeared in column\n");

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInBlock[j]);
        }
        printf(" - Appeared in block\n");
    }
}
__host__ void printBoard(int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d ", sudokuBoard[i * BOARD_SIZE + j]);
        }
        printf("\n");
    }
}
__host__ __device__ bool isBoardValid(int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[i * BOARD_SIZE + j] > 0)
                all[sudokuBoard[i * BOARD_SIZE + j] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] != 1)
                return false;
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[j * BOARD_SIZE + i] > 0)
                all[sudokuBoard[j * BOARD_SIZE + i] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] != 1)
                return false;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int all[BOARD_SIZE] = {0};
            for (int x = i * N; x < (i + 1) * N; x++)
            {
                for (int y = j * N; y < (j + 1) * N; y++)
                {
                    if (sudokuBoard[x * BOARD_SIZE + y] > 0)
                        all[sudokuBoard[x * BOARD_SIZE + y] - 1]++;
                }
            }
            for (int k = 0; k < BOARD_SIZE; k++)
                if (all[k] != 1)
                {
                    return false;
                }
        }
    }
    return true;
}
__device__ bool isBoardCorrect(const int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[i * BOARD_SIZE + j] > 0)
                all[sudokuBoard[i * BOARD_SIZE + j] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] > 1)
                return false;
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[j * BOARD_SIZE + i] > 0)
                all[sudokuBoard[j * BOARD_SIZE + i] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] > 1)
                return false;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int all[BOARD_SIZE] = {0};
            for (int x = i * N; x < (i + 1) * N; x++)
            {
                for (int y = j * N; y < (j + 1) * N; y++)
                {
                    if (sudokuBoard[x * BOARD_SIZE + y] > 0)
                        all[sudokuBoard[x * BOARD_SIZE + y] - 1]++;
                }
            }
            for (int k = 0; k < BOARD_SIZE; k++)
                if (all[k] > 1)
                {
                    return false;
                }
        }
    }
    return true;
}

__device__ void calculatePossibilites(const int *currentBoard, int *emptyCells, possibilitie *poss, int *possCount)
{
    int cell = -1;

    int appeardInRow[BOARD_SIZE] = {0};
    int appeardInColumn[BOARD_SIZE] = {0};
    int appeardInBlock[BOARD_SIZE] = {0};
    int emptyInAll[BOARD_SIZE] = {0};

    int indx = 0;
    int tmp = 0;

    for (int k = 0; k < *possCount; k++)
    {
        cell = emptyCells[k];

        for (int i = 0; i < BOARD_SIZE; i++)
        {
            int tmp = currentBoard[(cell / BOARD_SIZE) * BOARD_SIZE + i];
            if (tmp > 0)
            {
                appeardInRow[tmp - 1]++;
            }
        }

        for (int i = 0; i < BOARD_SIZE; i++)
        {

            int tmp = currentBoard[cell % BOARD_SIZE + i * BOARD_SIZE];
            if (tmp > 0)
            {
                appeardInColumn[tmp - 1]++;
            }
        }

        int firstCellOfBlock = ((cell / BOARD_SIZE) / N) * BOARD_SIZE * N + ((cell % BOARD_SIZE) / N) * N;
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                int tmp = currentBoard[firstCellOfBlock + i * BOARD_SIZE + j];
                if (tmp > 0)
                {
                    appeardInBlock[tmp - 1]++;
                }
            }
        }

        // Remember the possibilites
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (appeardInBlock[j] == 0 &&
                appeardInColumn[j] == 0 &&
                appeardInRow[j] == 0)
            {
                emptyInAll[j] = 1;
                tmp++;
            }
        }
        if (tmp > 0)
        {
            for (int i = 0; i < BOARD_SIZE; i++)
                poss[indx].poss[i] = emptyInAll[i];
            poss[indx].cell = cell;
            indx++;
        }

        // Reset arrays
        for (int i = 0; i < BOARD_SIZE; i++)
        {
            emptyInAll[i] = appeardInBlock[i] = appeardInColumn[i] = appeardInRow[i] = 0;
        }
        tmp = 0;
    }

    *possCount = indx;
}

__global__ void runSolver(const int *currentBoard, possibleBoard *possBoard)
{
    // Kerenl recives a board as an array size of CELL_COUNT
    // It generates valid boards that can be created
    // using given array and returns it to host.
    // If in board exists cells that are "sure" meaning only
    // one number can be inputed there, we only consider those
    // boards.

    // Count how many empty cells we have
    // i.e. how many possibilites can ther be
    int indx = 0;
    int emptyCells[CELL_COUNT] = {0};

    currentBoard += (CELL_COUNT)*threadIdx.x;
    possBoard += (BOARD_SIZE)*threadIdx.x;
    if (!isBoardCorrect(currentBoard))
        return;

    possibilitie *poss = new possibilitie[CELL_COUNT];
    for (int i = 0; i < CELL_COUNT; i++)
    {
        if (currentBoard[i] == 0)
        {
            emptyCells[indx] = i;
            indx++;
        }
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < CELL_COUNT; j++)
            possBoard[i].board[j] = currentBoard[j];

        possBoard[i].status = 0;
    }
    for (int i = 0; i < CELL_COUNT; i++)
    {
        poss[i].cell = 0;
        for (int j = 0; j < BOARD_SIZE; j++)
            poss[i].poss[j] = 0;
    }

    calculatePossibilites(currentBoard, (int *)emptyCells, poss, &indx);

    // We now have all possible otions that can be safely inputted into our
    // current board.

    int leastOption = 11, iWithLeastOptions = -1;

    for (int i = 0; i < indx; i++)
    {
        int tmp = 0;
        for (int j = 0; j < BOARD_SIZE; j++)
            tmp += poss[i].poss[j];
        if (tmp < leastOption && tmp > 0)
        {
            leastOption = tmp;
            iWithLeastOptions = i;
        }
    }

    int countOfBoards = 0;
    if (iWithLeastOptions > -1)
        for (int i = 0; i < BOARD_SIZE; i++)
        {
            if (poss[iWithLeastOptions].poss[i] == 1)
            {

                possBoard[countOfBoards].status = leastOption;
                possBoard[countOfBoards].board[poss[iWithLeastOptions].cell] = i + 1;
                countOfBoards++;
            }
        }

    delete[] poss;
}
__host__ int solveSudoku(const int *start_board)
{
    hipError_t cudaStatus;
    int *sudokuBoard = 0;
    int tmpSudokuBoard[CELL_COUNT * NUM_OF_THREADS];
    possibleBoard *poss_d = 0, *poss_h = 0;

    poss_h = new possibleBoard[BOARD_SIZE * NUM_OF_THREADS];
    auto cmp = [](possibleBoard left, possibleBoard right)
    { return (left.status) > (right.status); };
    std::priority_queue<possibleBoard, std::vector<possibleBoard>, decltype(cmp)> S(cmp);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&sudokuBoard, NUM_OF_THREADS * CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(sudokuBoard, start_board, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    printBoard((int *)start_board);
    runSolver<<<1, 1>>>(sudokuBoard, poss_d);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(poss_h, poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! Returned error code %d\n", cudaStatus);
        goto Error;
    }

    for (int i = 0; i < NUM_OF_THREADS; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (poss_h[i * BOARD_SIZE + j].status >= 1)
            {
#ifdef DEBUG_MODE
                printf("Possible board [THREAD: %d][POSS: %d]:\n", i + 1, j + 1);
                printBoard(poss_h[i * BOARD_SIZE + j].board);
#endif
                possibleBoard tmp;
                for (int k = 0; k < CELL_COUNT; k++)
                    tmp.board[k] = poss_h[i * BOARD_SIZE + j].board[k];
                tmp.status = poss_h[i * BOARD_SIZE + j].status;
                S.push(tmp);
            }
        }
    }

    while (!S.empty())
    {
        // Input new boards
        int indx = 0;
#ifdef DEBUG_MODE
        printf("%ld\n", S.size());
#endif
        for (; indx < NUM_OF_THREADS && !S.empty(); indx++)
        {
            possibleBoard tmp = S.top();
            S.pop();
#ifdef DEBUG_MODE
            printf("Running thread %02d with board:\n", indx + 1);
            printBoard(tmp.board);
#endif
            for (int j = 0; j < CELL_COUNT; j++)
            {
                tmpSudokuBoard[indx * CELL_COUNT + j] = tmp.board[j];
            }
        }

        // Copy memory and run kernel
        cudaStatus = hipMemcpy(sudokuBoard, tmpSudokuBoard, (NUM_OF_THREADS)*CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
#ifdef DEBUG_MODE
        printf("Running %02d threads\n", indx);
#endif
        runSolver<<<1, indx>>>(sudokuBoard, poss_d);

        // Fetch resoults
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy(poss_h, poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed! Returned error code %d\n", cudaStatus);
            goto Error;
        }

        // Add new boards to S
        for (int i = 0; i < indx; i++)
        {
            for (int j = 0; j < BOARD_SIZE; j++)
            {
                if (poss_h[i * BOARD_SIZE + j].status >= 1)
                {
#ifdef DEBUG_MODE
                    printf("Possible board [THREAD: %d][POSS: %d]:\n", i + 1, j + 1);
                    printBoard(poss_h[i * BOARD_SIZE + j].board);
#endif
                    if (isBoardValid(poss_h[i * BOARD_SIZE + j].board))
                    {
                        printf("SOLVED!\n");
                        printBoard(poss_h[i * BOARD_SIZE + j].board);
                        goto Error;
                    }

                    possibleBoard tmp;
                    for (int k = 0; k < CELL_COUNT; k++)
                        tmp.board[k] = poss_h[i * BOARD_SIZE + j].board[k];
                    tmp.status = poss_h[i * BOARD_SIZE + j].status;
                    S.push(tmp);
                }
            }
        }
    }

Error:
    hipFree(sudokuBoard);
    hipFree(poss_d);
    delete[] poss_h;

    return cudaStatus;
}

int main()
{
    // Easy
    // const int start_board[CELL_COUNT] =
    //     {
    //         3, 8, 6, 0, 0, 4, 7, 0, 0,
    //         0, 0, 9, 0, 0, 0, 2, 0, 0,
    //         0, 2, 0, 1, 0, 3, 8, 0, 5,
    //         0, 7, 8, 0, 3, 0, 6, 2, 0,
    //         0, 5, 2, 0, 0, 1, 0, 0, 4,
    //         9, 4, 0, 2, 7, 0, 0, 0, 0,
    //         2, 3, 0, 7, 4, 9, 5, 8, 6,
    //         8, 0, 0, 0, 1, 0, 4, 0, 0,
    //         4, 0, 0, 0, 0, 0, 0, 0, 2,
    //     };
    // Hard
    // const int start_board[CELL_COUNT] =
    //     {
    //         0, 0, 1, 0, 0, 0, 3, 6, 0,
    //         0, 0, 0, 0, 2, 0, 0, 0, 0,
    //         3, 0, 0, 0, 5, 6, 0, 8, 0,
    //         0, 0, 0, 9, 0, 0, 0, 0, 0,
    //         0, 4, 0, 0, 0, 0, 0, 0, 7,
    //         1, 0, 0, 0, 3, 8, 0, 5, 0,
    //         0, 0, 0, 1, 0, 0, 0, 9, 0,
    //         0, 0, 7, 0, 6, 9, 0, 0, 5,
    //         6, 0, 0, 2, 0, 0, 0, 0, 0,
    //     };
    // const int start_board[CELL_COUNT] =
    //     {
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //     };

    const int start_board1[CELL_COUNT] =
        {
            0,
            0,
            1,
            0,
            0,
            0,
            3,
            6,
            0,
            0,
            0,
            0,
            0,
            2,
            0,
            0,
            0,
            0,
            3,
            0,
            0,
            0,
            5,
            6,
            0,
            8,
            0,
            0,
            0,
            0,
            9,
            0,
            0,
            0,
            0,
            0,
            0,
            4,
            0,
            0,
            0,
            0,
            0,
            0,
            7,
            1,
            0,
            0,
            0,
            3,
            8,
            0,
            5,
            0,
            0,
            0,
            0,
            1,
            0,
            0,
            0,
            9,
            0,
            0,
            0,
            7,
            0,
            6,
            9,
            0,
            0,
            5,
            6,
            0,
            0,
            2,
            0,
            0,
            0,
            0,
            0,
        };
    const int start_board2[CELL_COUNT] =
        {
            3,
            8,
            6,
            0,
            0,
            4,
            7,
            0,
            0,
            0,
            0,
            9,
            0,
            0,
            0,
            2,
            0,
            0,
            0,
            2,
            0,
            1,
            0,
            3,
            8,
            0,
            5,
            0,
            7,
            8,
            0,
            3,
            0,
            6,
            2,
            0,
            0,
            5,
            2,
            0,
            0,
            1,
            0,
            0,
            4,
            9,
            4,
            0,
            2,
            7,
            0,
            0,
            0,
            0,
            2,
            3,
            0,
            7,
            4,
            9,
            5,
            8,
            6,
            8,
            0,
            0,
            0,
            1,
            0,
            4,
            0,
            0,
            4,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            2,
        };

    const int start_board3[CELL_COUNT] =
        {
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
        };

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    int tmp = solveSudoku((int *)start_board1);
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time elapsed = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    begin = std::chrono::steady_clock::now();
    tmp = solveSudoku((int *)start_board2);
    end = std::chrono::steady_clock::now();
    std::cout << "Time elapsed = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    return tmp;
}