#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

#include <stack>

#define N 3
#define BOARD_SIZE (N * N)
#define CELL_COUNT (BOARD_SIZE * BOARD_SIZE)

#define DEBUG_MODE

struct appeared
{
    int cell = -1;

    int appeardInRow[BOARD_SIZE] = {0};
    int appeardInColumn[BOARD_SIZE] = {0};
    int appeardInBlock[BOARD_SIZE] = {0};

} typedef appeared;

struct move
{
    int board[CELL_COUNT] = {0};
    int cell = -1;
    int possibilites[BOARD_SIZE] = {0};
    int moveMade = -1;

    move(int *currentBoard, int cell, int *possibilites, int move)
    {
        for (int i = 0; i < CELL_COUNT; i++)
            this->board[i] = currentBoard[i];

        for (int i = 0; i < BOARD_SIZE; i++)
            this->possibilites[i] = possibilites[i];

        this->cell = cell;
        this->moveMade = move;
    }
} typedef move;

__global__ void
fillEmpty(const int *sudokuBoard, const int *targetCell, appeared *app);

__host__ void printBoardInfo(appeared *app)
{
    for (int i = 0; i < CELL_COUNT; i++)
    {

        printf("[%d]TARGET CELL - %d\n", i, app[i].cell);

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInRow[j]);
        }
        printf(" - Appeared in row\n");

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInColumn[j]);
        }
        printf(" - Appeared in column\n");

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInBlock[j]);
        }
        printf(" - Appeared in block\n");
    }
}
__host__ void printBoard(int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d ", sudokuBoard[i * BOARD_SIZE + j]);
        }
        printf("\n");
    }
}
__host__ bool isBoardValid(int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[i * BOARD_SIZE + j] > 0)
                all[sudokuBoard[i * BOARD_SIZE + j] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] != 1)
                return false;
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[j * BOARD_SIZE + i] > 0)
                all[sudokuBoard[j * BOARD_SIZE + i] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] != 1)
                return false;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int all[BOARD_SIZE] = {0};
            for (int x = i * N; x < (i + 1) * N; x++)
            {
                for (int y = j * N; y < (j + 1) * N; y++)
                {
                    if (sudokuBoard[x * BOARD_SIZE + y] > 0)
                        all[sudokuBoard[x * BOARD_SIZE + y] - 1]++;
                }
            }
            for (int k = 0; k < BOARD_SIZE; k++)
                if (all[k] != 1)
                {
                    return false;
                }
        }
    }
    return true;
}

__host__ void solve(int indx, int *sudokuBoard, int *targetCell, appeared *app, appeared *calculated, const int *start_board, hipError_t &cudaStatus)
{
    std::stack<move> S;

    int currentBoard[CELL_COUNT] = {0};
    int empty_cells[CELL_COUNT] = {-1};
    for (int i = 0; i < CELL_COUNT; i++)
        currentBoard[i] = start_board[i];
    while (!isBoardValid(currentBoard))
    {
        // Calucate notes
        fillEmpty<<<1, indx>>>(sudokuBoard, targetCell, app);

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        }

        // cudaDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        }
        cudaStatus = hipMemcpy(calculated, app, CELL_COUNT * sizeof(appeared), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed!");
        }

        // Check notes, where we have posibility to enter something in cell

#ifdef DEBUG_MODE
        printf("Current board is ");
        printf((isBoardValid(currentBoard) ? "valid" : "invalid"));
        printf(":\n");
        printBoard(currentBoard);
        // printBoardInfo(calculated);
#endif

        int iWithLeastOptions = -1;
        int optionsWithI = -1;
        int emptyInAll[CELL_COUNT][BOARD_SIZE] = {0};

        for (int i = 0; i < indx; i++)
        {
            int tmp = 0;
            for (int j = 0; j < BOARD_SIZE; j++)
            {
                if (calculated[i].appeardInBlock[j] == 0 &&
                    calculated[i].appeardInColumn[j] == 0 &&
                    calculated[i].appeardInRow[j] == 0)
                {
                    emptyInAll[i][j] = 1;
                    tmp++;
                }
            }

            if ((iWithLeastOptions < 0 || optionsWithI > tmp) && tmp > 0)
            {
                iWithLeastOptions = i;
                optionsWithI = tmp;
            }
        }

        // Do move

        printf("Least options: %d; For cell: %d\n", optionsWithI, calculated[iWithLeastOptions].cell);
        if (optionsWithI == 1)
        {
            for (int i = 0; i < BOARD_SIZE; i++)
                if (emptyInAll[iWithLeastOptions][i] == 1)
                {
                    emptyInAll[iWithLeastOptions][i] = 0;
                    move m = move((int *)currentBoard, calculated[iWithLeastOptions].cell, (int *)emptyInAll[iWithLeastOptions], i);
                    currentBoard[calculated[iWithLeastOptions].cell] = i + 1;
                    S.push(m);
                    break;
                }
        }
        else if (optionsWithI > 1)
        {
            // Input random number and check if board is valid
            for (int i = 0; i < BOARD_SIZE; i++)
            {
                if (emptyInAll[iWithLeastOptions][i] == 1)
                {
                    emptyInAll[iWithLeastOptions][i] = 0;
                    currentBoard[calculated[iWithLeastOptions].cell] = i + 1;
                    move m = move((int *)currentBoard, calculated[iWithLeastOptions].cell, (int *)emptyInAll[iWithLeastOptions], i);
                    S.push(m);
                    break;
                }
            }
        }
        else if (!isBoardValid(currentBoard))
        {
            // Board is broken, we need to back up
            printf("We are backing up\n");
            bool foundMove = false;
            while (!S.empty() && !foundMove)
            {
                move m = S.top();
                S.pop();
                int numOfPossibilites = 0;
                for (int i = 0; i < BOARD_SIZE; i++)
                    numOfPossibilites += m.possibilites[i];

                // We do not have an option to diverge from this state
                if (numOfPossibilites == 0)
                {
                    printf("Move with no options\n");
                    continue;
                }

                for (int i = 0; i < BOARD_SIZE; i++)
                {
                    if (m.possibilites[i] == 1)
                    {
                        // We have a possiblitie to diverge
                        m.possibilites[i] = 0;
                        m.moveMade = i;
                        for (int j = 0; j < CELL_COUNT; j++)
                            currentBoard[j] = m.board[j];
                        currentBoard[m.cell] = i + 1;
                        S.push(m);
                        printf("Move with   options - cell: %d set to: %d\n", m.cell, i + 1);
                        foundMove = true;
                        break;
                    }
                }
            }
            if (S.empty())
                return;
        }

        // Prepear next step

#ifdef DEBUG_MODE
        printf("Board with changes is ");
        printf((isBoardValid(currentBoard) ? "valid" : "invalid"));
        printf(":\n");
        printBoard(currentBoard);
#endif

        for (int i = 0; i < CELL_COUNT; i++)
        {
            cudaStatus = hipMemcpy((sudokuBoard + i * CELL_COUNT), currentBoard, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "hipMemcpy failed!");
            }
        }

        indx = 0;
        for (int i = 0; i < CELL_COUNT; i++)
            if (currentBoard[i] == 0)
            {
                empty_cells[indx] = i;
                indx++;
            }

        cudaStatus = hipMemcpy(targetCell, empty_cells, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed!");
        }
    }
}

__global__ void fillEmpty(const int *sudokuBoard, const int *targetCell, appeared *app)
{

    sudokuBoard += threadIdx.x * CELL_COUNT;
    app[threadIdx.x].cell = targetCell[threadIdx.x];

    for (int i = 0; i < BOARD_SIZE; i++)
    {
        app[threadIdx.x].appeardInBlock[i] = app[threadIdx.x].appeardInColumn[i] = app[threadIdx.x].appeardInRow[i] = 0;
    }

    // Calculate notes -- if it turns out there is only one possiblity - insert it

    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int tmp = sudokuBoard[(app[threadIdx.x].cell / BOARD_SIZE) * BOARD_SIZE + i];
        if (tmp > 0)
        {
            app[threadIdx.x].appeardInRow[tmp - 1]++;
        }
    }

    for (int i = 0; i < BOARD_SIZE; i++)
    {

        int tmp = sudokuBoard[app[threadIdx.x].cell % BOARD_SIZE + i * BOARD_SIZE];
        if (tmp > 0)
        {
            app[threadIdx.x].appeardInColumn[tmp - 1]++;
        }
    }

    int firstCellOfBlock = ((app[threadIdx.x].cell / BOARD_SIZE) / N) * BOARD_SIZE * N + ((app[threadIdx.x].cell % BOARD_SIZE) / N) * N;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int tmp = sudokuBoard[firstCellOfBlock + i * BOARD_SIZE + j];
            if (tmp > 0)
            {
                app[threadIdx.x].appeardInBlock[tmp - 1]++;
            }
        }
    }
}

__host__ int solveSudoku(const int *start_board, int *sudokuBoard, int *targetCell, appeared *app)
{
    hipError_t cudaStatus;

    int empty_cells[CELL_COUNT] = {-1};

    int indx = 0;
    for (int i = 0; i < CELL_COUNT; i++)
        if (start_board[i] == 0)
        {
            empty_cells[indx] = i;
            indx++;
        }

    appeared calculated[CELL_COUNT];

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **)&sudokuBoard, CELL_COUNT * CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&targetCell, CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&app, CELL_COUNT * sizeof(appeared));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    for (int i = 0; i < CELL_COUNT; i++)
    {
        cudaStatus = hipMemcpy((sudokuBoard + i * CELL_COUNT), start_board, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

    cudaStatus = hipMemcpy(targetCell, empty_cells, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    solve(indx, sudokuBoard, targetCell, app, calculated, start_board, cudaStatus);

Error:
    hipFree(sudokuBoard);
    hipFree(targetCell);
    hipFree(app);

    return cudaStatus;
}

int main()
{

    // const int start_board[CELL_COUNT] =
    //     {
    //         3, 8, 6, 0, 0, 4, 7, 0, 0,
    //         0, 0, 9, 0, 0, 0, 2, 0, 0,
    //         0, 2, 0, 1, 0, 3, 8, 0, 5,
    //         0, 7, 8, 0, 3, 0, 6, 2, 0,
    //         0, 5, 2, 0, 0, 1, 0, 0, 4,
    //         9, 4, 0, 2, 7, 0, 0, 0, 0,
    //         2, 3, 0, 7, 4, 9, 5, 8, 6,
    //         8, 0, 0, 0, 1, 0, 4, 0, 0,
    //         4, 0, 0, 0, 0, 0, 0, 0, 2,
    //     };
    // const int start_board[CELL_COUNT] =
    //     {
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //         0, 0, 0, 0, 0, 0, 0, 0, 0,
    //     };

    const int start_board[CELL_COUNT] =
        {
            3,
            8,
            6,
            0,
            0,
            4,
            7,
            0,
            0,
            0,
            0,
            9,
            0,
            0,
            0,
            2,
            0,
            0,
            0,
            2,
            0,
            1,
            0,
            3,
            8,
            0,
            5,
            0,
            7,
            8,
            0,
            3,
            0,
            6,
            2,
            0,
            0,
            5,
            2,
            0,
            0,
            1,
            0,
            0,
            4,
            9,
            4,
            0,
            2,
            7,
            0,
            0,
            0,
            0,
            2,
            3,
            0,
            7,
            4,
            9,
            5,
            8,
            6,
            8,
            0,
            0,
            0,
            1,
            0,
            4,
            0,
            0,
            4,
            0,
            0,
            0,
            0,
            0,
            0,
            0,
            2,
        };

    int *sudokuBoard = 0;
    int *targetCell = 0;
    appeared *app = 0;

    return solveSudoku((int *)start_board, sudokuBoard, targetCell, app);
}