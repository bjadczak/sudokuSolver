#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>

#include <stack>
#include <queue>
#include <chrono>

#define N 3
#define BOARD_SIZE (N * N)
#define CELL_COUNT (BOARD_SIZE * BOARD_SIZE)
#define NUM_OF_THREADS 1024
#define NUM_OF_BLOCKS 10
#define NUM_OF_KERNELS (NUM_OF_THREADS * NUM_OF_BLOCKS)

//#define DEBUG_MODE

struct possibilitie
{
    int poss[BOARD_SIZE] = {0};
    int cell = -1;

} typedef possibilitie;
struct possibleBoard
{
    int board[CELL_COUNT] = {0};
    int status = 0;

} typedef possibleBoard;
struct board
{
    int board[CELL_COUNT] = {0};

} typedef board;

__host__ void printBoard(int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d ", sudokuBoard[i * BOARD_SIZE + j]);
        }
        printf("\n");
    }
}
__host__ __device__ bool isBoardValid(int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[i * BOARD_SIZE + j] > 0)
                all[sudokuBoard[i * BOARD_SIZE + j] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] != 1)
                return false;
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[j * BOARD_SIZE + i] > 0)
                all[sudokuBoard[j * BOARD_SIZE + i] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] != 1)
                return false;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int all[BOARD_SIZE] = {0};
            for (int x = i * N; x < (i + 1) * N; x++)
            {
                for (int y = j * N; y < (j + 1) * N; y++)
                {
                    if (sudokuBoard[x * BOARD_SIZE + y] > 0)
                        all[sudokuBoard[x * BOARD_SIZE + y] - 1]++;
                }
            }
            for (int k = 0; k < BOARD_SIZE; k++)
                if (all[k] != 1)
                {
                    return false;
                }
        }
    }
    return true;
}
__device__ bool isBoardCorrect(const int *sudokuBoard)
{
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[i * BOARD_SIZE + j] > 0)
                all[sudokuBoard[i * BOARD_SIZE + j] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] > 1)
                return false;
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int all[BOARD_SIZE] = {0};
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (sudokuBoard[j * BOARD_SIZE + i] > 0)
                all[sudokuBoard[j * BOARD_SIZE + i] - 1]++;
        }
        for (int j = 0; j < BOARD_SIZE; j++)
            if (all[j] > 1)
                return false;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int all[BOARD_SIZE] = {0};
            for (int x = i * N; x < (i + 1) * N; x++)
            {
                for (int y = j * N; y < (j + 1) * N; y++)
                {
                    if (sudokuBoard[x * BOARD_SIZE + y] > 0)
                        all[sudokuBoard[x * BOARD_SIZE + y] - 1]++;
                }
            }
            for (int k = 0; k < BOARD_SIZE; k++)
                if (all[k] > 1)
                {
                    return false;
                }
        }
    }
    return true;
}

__device__ void calculatePossibilites(const int *currentBoard, int *emptyCells, possibilitie *poss, int *possCount)
{
    int cell = -1;

    int appeardInRow[BOARD_SIZE] = {0};
    int appeardInColumn[BOARD_SIZE] = {0};
    int appeardInBlock[BOARD_SIZE] = {0};
    int emptyInAll[BOARD_SIZE] = {0};

    int indx = 0;
    int tmp = 0;

    for (int k = 0; k < *possCount; k++)
    {
        cell = emptyCells[k];

        for (int i = 0; i < BOARD_SIZE; i++)
        {
            int tmp = currentBoard[(cell / BOARD_SIZE) * BOARD_SIZE + i];
            if (tmp > 0)
            {
                appeardInRow[tmp - 1]++;
            }
        }

        for (int i = 0; i < BOARD_SIZE; i++)
        {

            int tmp = currentBoard[cell % BOARD_SIZE + i * BOARD_SIZE];
            if (tmp > 0)
            {
                appeardInColumn[tmp - 1]++;
            }
        }

        int firstCellOfBlock = ((cell / BOARD_SIZE) / N) * BOARD_SIZE * N + ((cell % BOARD_SIZE) / N) * N;
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                int tmp = currentBoard[firstCellOfBlock + i * BOARD_SIZE + j];
                if (tmp > 0)
                {
                    appeardInBlock[tmp - 1]++;
                }
            }
        }

        // Remember the possibilites
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (appeardInBlock[j] == 0 &&
                appeardInColumn[j] == 0 &&
                appeardInRow[j] == 0)
            {
                emptyInAll[j] = 1;
                tmp++;
            }
        }
        if (tmp > 0)
        {
            for (int i = 0; i < BOARD_SIZE; i++)
                poss[indx].poss[i] = emptyInAll[i];
            poss[indx].cell = cell;
            indx++;
        }

        // Reset arrays
        for (int i = 0; i < BOARD_SIZE; i++)
        {
            emptyInAll[i] = appeardInBlock[i] = appeardInColumn[i] = appeardInRow[i] = 0;
        }
        tmp = 0;
    }

    *possCount = indx;
}

__global__ void runSolver(const int *currentBoard, possibleBoard *possBoard)
{
    // Kerenl recives a board as an array size of CELL_COUNT
    // It generates valid boards that can be created
    // using given array and returns it to host.
    // If in board exists cells that are "sure" meaning only
    // one number can be inputed there, we only consider those
    // boards.

    // Count how many empty cells we have
    // i.e. how many possibilites can ther be
    int indx = 0;
    int emptyCells[CELL_COUNT] = {0};

    int numOfEmptyCells = 0;

    currentBoard += (CELL_COUNT)*threadIdx.x;
    possBoard += (BOARD_SIZE)*threadIdx.x;
    if (!isBoardCorrect(currentBoard))
        return;

    possibilitie *poss = new possibilitie[CELL_COUNT];
    for (int i = 0; i < CELL_COUNT; i++)
    {
        if (currentBoard[i] == 0)
        {
            emptyCells[indx] = i;
            indx++;
        }
    }
    for (int i = 0; i < BOARD_SIZE; i++)
    {
        for (int j = 0; j < CELL_COUNT; j++)
            possBoard[i].board[j] = currentBoard[j];

        possBoard[i].status = 0;
    }
    for (int i = 0; i < CELL_COUNT; i++)
    {
        poss[i].cell = 0;
        for (int j = 0; j < BOARD_SIZE; j++)
            poss[i].poss[j] = 0;
    }

    numOfEmptyCells = indx;
    calculatePossibilites(currentBoard, (int *)emptyCells, poss, &indx);

    // We now have all possible otions that can be safely inputted into our
    // current board.

    int leastOption = 11, iWithLeastOptions = -1;

    for (int i = 0; i < indx; i++)
    {
        int tmp = 0;
        for (int j = 0; j < BOARD_SIZE; j++)
            tmp += poss[i].poss[j];
        if (tmp < leastOption && tmp > 0)
        {
            leastOption = tmp;
            iWithLeastOptions = i;
        }
    }

    int countOfBoards = 0;
    if (iWithLeastOptions > -1)
        for (int i = 0; i < BOARD_SIZE; i++)
        {
            if (poss[iWithLeastOptions].poss[i] == 1)
            {

                possBoard[countOfBoards].status = leastOption + numOfEmptyCells;
                possBoard[countOfBoards].board[poss[iWithLeastOptions].cell] = i + 1;
                countOfBoards++;
            }
        }

    delete[] poss;
}
__host__ int solveSudoku(int *start_board)
{
    hipError_t cudaStatus;
    int *sudokuBoard = 0;
    int tmpSudokuBoard[CELL_COUNT * NUM_OF_THREADS];
    possibleBoard *poss_d = 0, *poss_h = 0;

    poss_h = new possibleBoard[BOARD_SIZE * NUM_OF_THREADS];
    auto cmp = [](possibleBoard left, possibleBoard right)
    { return (left.status) > (right.status); };
    std::priority_queue<possibleBoard, std::vector<possibleBoard>, decltype(cmp)> S(cmp);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&sudokuBoard, NUM_OF_THREADS * CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(sudokuBoard, start_board, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    printBoard((int *)start_board);
    runSolver<<<1, 1>>>(sudokuBoard, poss_d);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(poss_h, poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! Returned error code %d\n", cudaStatus);
        goto Error;
    }

    for (int i = 0; i < NUM_OF_THREADS; i++)
    {
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (poss_h[i * BOARD_SIZE + j].status >= 1)
            {
#ifdef DEBUG_MODE
                printf("Possible board [THREAD: %d][POSS: %d]:\n", i + 1, j + 1);
                printBoard(poss_h[i * BOARD_SIZE + j].board);
#endif
                possibleBoard tmp;
                for (int k = 0; k < CELL_COUNT; k++)
                    tmp.board[k] = poss_h[i * BOARD_SIZE + j].board[k];
                tmp.status = poss_h[i * BOARD_SIZE + j].status;
                S.push(tmp);
            }
        }
    }

    while (!S.empty())
    {
        // Input new boards
        int indx = 0;
#ifdef DEBUG_MODE
        printf("%ld\n", S.size());
#endif
        for (; indx < NUM_OF_THREADS && !S.empty(); indx++)
        {
            possibleBoard tmp = S.top();
            S.pop();
#ifdef DEBUG_MODE
            printf("Running thread %02d with board:\n", indx + 1);
            printBoard(tmp.board);
#endif
            for (int j = 0; j < CELL_COUNT; j++)
            {
                tmpSudokuBoard[indx * CELL_COUNT + j] = tmp.board[j];
            }
        }

        // Copy memory and run kernel
        cudaStatus = hipMemcpy(sudokuBoard, tmpSudokuBoard, (NUM_OF_THREADS)*CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
#ifdef DEBUG_MODE
        printf("Running %02d threads\n", indx);
#endif
        runSolver<<<1, indx>>>(sudokuBoard, poss_d);

        // Fetch resoults
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy(poss_h, poss_d, NUM_OF_THREADS * BOARD_SIZE * sizeof(possibleBoard), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed! Returned error code %d\n", cudaStatus);
            goto Error;
        }

        // Add new boards to S
        for (int i = 0; i < indx; i++)
        {
            for (int j = 0; j < BOARD_SIZE; j++)
            {
                if (poss_h[i * BOARD_SIZE + j].status >= 1)
                {
#ifdef DEBUG_MODE
                    printf("Possible board [THREAD: %d][POSS: %d]:\n", i + 1, j + 1);
                    printBoard(poss_h[i * BOARD_SIZE + j].board);
#endif
                    if (isBoardValid(poss_h[i * BOARD_SIZE + j].board))
                    {
                        printf("SOLVED!\n");
                        printBoard(poss_h[i * BOARD_SIZE + j].board);
                        goto Error;
                    }

                    possibleBoard tmp;
                    for (int k = 0; k < CELL_COUNT; k++)
                        tmp.board[k] = poss_h[i * BOARD_SIZE + j].board[k];
                    tmp.status = poss_h[i * BOARD_SIZE + j].status;
                    S.push(tmp);
                }
            }
        }
    }

Error:
    hipFree(sudokuBoard);
    hipFree(poss_d);
    delete[] poss_h;

    return cudaStatus;
}

__host__ void loadBoard(int *board, std::ifstream &inFile)
{
    std::string line;
    for (int i = 0; i < BOARD_SIZE && getline(inFile, line); i++)
    {
        if (line.length() != BOARD_SIZE)
        {
            i--;
        }
        else
            for (int j = 0; j < BOARD_SIZE; j++)
            {
                board[i * BOARD_SIZE + j] = line[j] - '0';
            }
    }
}

int main(int argc, char **argv)
{

    std::ifstream inFS;
    std::string fileName;

    std::cout << "Input file: ";
    std::cin >> fileName;

    int *board = new int[CELL_COUNT];

    std::chrono::steady_clock::time_point begin;
    std::chrono::steady_clock::time_point end;
    int tmp;

    inFS.open(fileName);

    while (!inFS.eof())
    {
        loadBoard(board, inFS);
        begin = std::chrono::steady_clock::now();
        tmp = solveSudoku(board);
        end = std::chrono::steady_clock::now();
        if (tmp != 0)
            return tmp;
        std::cout << "Time elapsed = " << std::chrono::duration_cast<std::chrono::milliseconds>(end - begin).count() << "[ms]" << std::endl;
    }

    inFS.close();
    delete[] board;

    return tmp;
}