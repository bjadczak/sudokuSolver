#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

#include <queue>

#define N 3
#define BOARD_SIZE (N * N)
#define CELL_COUNT (BOARD_SIZE * BOARD_SIZE)

#define DEBUG_MODE

struct appeared
{
    int cell = -1;

    int appeardInRow[BOARD_SIZE] = {0};
    int appeardInColumn[BOARD_SIZE] = {0};
    int appeardInBlock[BOARD_SIZE] = {0};

} typedef appeared;

__global__ void fillEmpty(const int *sudokuBoard, const int *targetCell, appeared *app);

__host__ void printBoard(appeared *app)
{
    for (int i = 0; i < CELL_COUNT; i++)
    {

        printf("[%d]TARGET CELL - %d\n", i, app[i].cell);

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInRow[j]);
        }
        printf(" - Appeared in row\n");

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInColumn[j]);
        }
        printf(" - Appeared in column\n");

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d", app[i].appeardInBlock[j]);
        }
        printf(" - Appeared in block\n");
    }
}

__host__ void solve(int indx, int *sudokuBoard, int *targetCell, appeared *app, appeared *calculated, hipError_t &cudaStatus)
{
    std::queue<int[CELL_COUNT]> Q;

    // Calucate notes
    fillEmpty<<<1, indx>>>(sudokuBoard, targetCell, app);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
    cudaStatus = hipMemcpy(calculated, app, CELL_COUNT * sizeof(appeared), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // Check notes, where we have posibility to enter something in cell

    int iWithLeastOptions = -1;
    int optionsWithI = -1;
    int emptyInAll[CELL_COUNT][BOARD_SIZE] = {0};

    for (int i = 0; i < indx; i++)
    {
        int tmp = 0;
        for (int j = 0; j < BOARD_SIZE; j++)
        {
            if (calculated[i].appeardInBlock[j] == 0 &&
                calculated[i].appeardInColumn[j] == 0 &&
                calculated[i].appeardInRow[j] == 0)
            {
                emptyInAll[i][j] = 1;
                tmp++;
            }
        }

        if (iWithLeastOptions < 0 || optionsWithI > tmp)
        {
            iWithLeastOptions = i;
            optionsWithI = tmp;
        }
#ifdef DEBUG_MODE
        printf("[%02d] Cell: %2d; Possible to input: ", i, calculated[i].cell);

        for (int j = 0; j < BOARD_SIZE; j++)
        {
            printf("%d ", emptyInAll[i][j]);
        }
        printf("\n");
#endif
    }

    printf("Least options: %d; For cell: %d\n", optionsWithI, calculated[iWithLeastOptions].cell);

#ifdef DEBUG_MODE
    printBoard(calculated);
#endif
}

__global__ void fillEmpty(const int *sudokuBoard, const int *targetCell, appeared *app)
{

    sudokuBoard += threadIdx.x * CELL_COUNT;
    app[threadIdx.x].cell = targetCell[threadIdx.x];

    // Calculate notes -- if it turns out there is only one possiblity - insert it

    for (int i = 0; i < BOARD_SIZE; i++)
    {
        int tmp = sudokuBoard[app[threadIdx.x].cell / BOARD_SIZE + i];
        if (tmp > 0)
        {
            app[threadIdx.x].appeardInRow[tmp - 1]++;
        }
    }

    for (int i = 0; i < BOARD_SIZE; i++)
    {

        int tmp = sudokuBoard[app[threadIdx.x].cell % BOARD_SIZE + i * BOARD_SIZE];
        if (tmp > 0)
        {
            app[threadIdx.x].appeardInColumn[tmp - 1]++;
        }
    }

    int firstCellOfBlock = ((app[threadIdx.x].cell / BOARD_SIZE) / N) * BOARD_SIZE * N + ((app[threadIdx.x].cell % BOARD_SIZE) / N) * N;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int tmp = sudokuBoard[firstCellOfBlock + i * BOARD_SIZE + j];
            if (tmp > 0)
            {
                app[threadIdx.x].appeardInBlock[tmp - 1]++;
            }
        }
    }
#ifdef DEBUG_MODE
    // Check what notes we have

    int numOfZerosRow = 0;
    int numOfZerosColumn = 0;
    int numOfZerosBlock = 0;

    for (int i = 0; i < BOARD_SIZE; i++)
    {
        if (app[threadIdx.x].appeardInRow[i] == 0)
            numOfZerosRow++;
        if (app[threadIdx.x].appeardInColumn[i] == 0)
            numOfZerosColumn++;
        if (app[threadIdx.x].appeardInBlock[i] == 0)
            numOfZerosBlock++;
    }

    printf("TARGET CELL - %d (id: %d); number of zeros row: %d; column: %d; block: %d\n", app[threadIdx.x].cell + 1, threadIdx.x, numOfZerosRow, numOfZerosColumn, numOfZerosBlock);
#endif
    __syncthreads();
}

__host__ int solveSudoku(const int *start_board, int *sudokuBoard, int *targetCell, appeared *app)
{
    hipError_t cudaStatus;

    int empty_cells[CELL_COUNT] = {-1};

    int indx = 0;
    for (int i = 0; i < CELL_COUNT; i++)
        if (start_board[i] == 0)
        {
            empty_cells[indx] = i;
            indx++;
        }

    appeared calculated[CELL_COUNT];

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **)&sudokuBoard, CELL_COUNT * CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&targetCell, CELL_COUNT * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **)&app, CELL_COUNT * sizeof(appeared));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    for (int i = 0; i < CELL_COUNT; i++)
    {
        cudaStatus = hipMemcpy((sudokuBoard + i * CELL_COUNT), start_board, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

    cudaStatus = hipMemcpy(targetCell, empty_cells, CELL_COUNT * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    solve(indx, sudokuBoard, targetCell, app, calculated, cudaStatus);

Error:
    hipFree(sudokuBoard);
    hipFree(targetCell);
    hipFree(app);

    return cudaStatus;
}

int main()
{

    // const int start_board[CELL_COUNT] =
    //     {
    //         3, 0, 0, 8, 0, 1, 0, 0, 2,
    //         2, 0, 1, 0, 3, 0, 6, 0, 4,
    //         0, 0, 0, 0, 1, 0, 0, 0, 0,
    //         8, 0, 9, 0, 0, 0, 1, 0, 6,
    //         0, 6, 0, 0, 0, 0, 0, 5, 0,
    //         7, 0, 2, 0, 0, 0, 4, 0, 9,
    //         0, 0, 0, 5, 0, 9, 0, 0, 0,
    //         9, 0, 4, 0, 8, 0, 7, 0, 5,
    //         6, 0, 0, 0, 0, 7, 0, 0, 3,
    //     };

    const int start_board[CELL_COUNT] =
        {
            3,
            0,
            0,
            8,
            0,
            1,
            0,
            0,
            2,
            2,
            0,
            1,
            0,
            3,
            0,
            6,
            0,
            4,
            0,
            0,
            0,
            0,
            1,
            0,
            0,
            0,
            0,
            8,
            0,
            9,
            0,
            0,
            0,
            1,
            0,
            6,
            0,
            6,
            0,
            0,
            0,
            0,
            0,
            5,
            0,
            7,
            0,
            2,
            0,
            0,
            0,
            4,
            0,
            9,
            0,
            0,
            0,
            5,
            0,
            9,
            0,
            0,
            0,
            9,
            0,
            4,
            0,
            8,
            0,
            7,
            0,
            5,
            6,
            0,
            0,
            0,
            0,
            7,
            0,
            0,
            3,
        };

    int *sudokuBoard = 0;
    int *targetCell = 0;
    appeared *app = 0;

    return solveSudoku((int *)start_board, sudokuBoard, targetCell, app);
}